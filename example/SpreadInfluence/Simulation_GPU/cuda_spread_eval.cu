#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <unistd.h>
#include <string.h>
#include <igraph.h>
#include <hip/hip_runtime.h>
#include <time.h>

__global__ void actv(int *s, int *v, int *a, int *a_index,int *adj,int *d_result,int num_v,int i){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index<num_v){
		if(i==0){
			if( s[index] >= v[index]){
			d_result[index]=1;
			}
		}else{
			if(a[index]==1){
				d_result[index]=1;
			}else{
				int sv = s[index];
				int j = a_index[index];
				while(adj[j]!=-1){
					if (a[adj[j]]==1){
						sv++;
					}
					j++;
				}
				if(sv>=v[index]){
					d_result[index]=1;
				}
			}
		}
		
	}
}

char** split(char *string, char *del){
	char **str_res=(char**)malloc(1);
	char *token = strtok(string,del);
	int count=0;
	while (token!=NULL) {
		str_res = (char**) realloc (str_res, sizeof (char*) * ++count);
		str_res[count-1] = token;
		token = strtok(NULL,del);
	}
	return str_res;
}

int main(int argc, char *argv[])
{

	if(argc <3){
		printf("Error: missing input parameter\n run: ./gpu_spread <graph_filepath> <influence sequence> <T>");
		exit(1);
	}
	/* code */
	igraph_t graph;
	FILE *fd;
	igraph_vs_t vs,vs_all;
	igraph_vit_t vit,vit_all;

	char **inf_seq, **trs_seq;
	int T, num_v,num_e;
	//igraph_integer_t max_degree;
	//igraph_matrix_t adjm;

	fd = fopen(argv[1],"r");

	igraph_read_graph_ncol(&graph,fd,NULL,1,IGRAPH_ADD_WEIGHTS_NO,IGRAPH_UNDIRECTED);
	//igraph_to_undirected(&graph, IGRAPH_TO_UNDIRECTED_COLLAPSE,0);
	fclose(fd);

	inf_seq = split(argv[2],",");
	//trs_seq = split(argv[3],",");
	T=atoi(argv[3]);

	num_v =igraph_vcount(&graph);
	num_e =igraph_ecount(&graph);

	int adj_index[num_v];
	int *adj=(int*) malloc(sizeof(int));

	int size=0;
	int index=0;
	igraph_vs_all(&vs_all);
	igraph_vit_create(&graph,vs_all,&vit_all);
	while (!IGRAPH_VIT_END(vit_all)) {
		adj_index[index]=size;
		igraph_vs_adj(&vs,IGRAPH_VIT_GET(vit_all),IGRAPH_ALL);
		igraph_vit_create(&graph,vs,&vit);
		while (!IGRAPH_VIT_END(vit)) {
			adj=(int*) realloc(adj,(size+1)*sizeof(int));
			adj[size]=IGRAPH_VIT_GET(vit);
			size++;
			IGRAPH_VIT_NEXT(vit);
		}
		adj=(int*) realloc(adj,(size+1)*sizeof(int));
		adj[size]=-1;
		size++;
		igraph_vit_destroy(&vit);
		igraph_vs_destroy(&vs);
		index++;
		IGRAPH_VIT_NEXT(vit_all);
	}
	igraph_vit_destroy(&vit_all);
	igraph_vs_destroy(&vs_all);


	int inf_v[num_v]; // vertex's influence values
	int trs_v[num_v]; // vertex's threshold values
	int act_v[num_v]; // vertex's activation values
	int res[num_v];

	int *d_inf_v;
	int *d_trs_v;
	int *d_act_v;
	int *d_index;
	int *d_adjlist;
	int *d_result;

	for (int i = 0; i < num_v; i++)
	{
		inf_v[i]=0;
		trs_v[i]=0;
		act_v[i]=0;
		res[i]=0;
	}

	srand(time(NULL));
	
	//char **inf_seq_split = split(inf_seq,",");
	for (int i = 0; i < num_v; ++i)
	{
		inf_v[i]=atoi(inf_seq[i]);
	}
	/*
	for (int i = 0; i < num_v; ++i)
	{
		trs_v[i]=atoi(trs_seq[i]);
	}*/

	for (int i = 0; i < T; i++)
	{
		int r = rand()%num_v+0;
		igraph_vs_adj(&vs,r,IGRAPH_ALL);
		igraph_vit_create(&graph,vs,&vit);
		if(trs_v[r]<IGRAPH_VIT_SIZE(vit)){
			trs_v[r]++;
		}
		igraph_vit_destroy(&vit);
		igraph_vs_destroy(&vs);

	}		
	

	//printf("numero di vertici %d e numero di archi %d\n",num_v,num_e);

	/* CALCULATE A0*/

	hipMalloc((void **) &d_inf_v,num_v*sizeof(int));
	hipMalloc((void **) &d_trs_v,num_v*sizeof(int));
	hipMalloc((void **) &d_act_v,num_v*sizeof(int));
	hipMalloc((void **) &d_index,num_v*sizeof(int));
	hipMalloc((void **) &d_adjlist,size*sizeof(int));
	hipMalloc((void **) &d_result,num_v*sizeof(int));

	hipMemcpy(d_inf_v,inf_v,num_v*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_trs_v,trs_v,num_v*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_act_v,act_v,num_v*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_result,res,num_v*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_index,adj_index,index*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_adjlist,adj,size*sizeof(int),hipMemcpyHostToDevice);


	int thread = 8;
	int block;
	if(num_v%thread==0){
		block = (num_v/thread);
	}else{
		block = (num_v/thread)+1;
	}


	int num_act_v=0;

	actv<<<block,thread>>>(d_inf_v,d_trs_v,d_act_v,d_index,d_adjlist,d_result,num_v,0);
	hipMemcpy(res,d_result,num_v*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0;i<num_v;i++){
		if(res[i]==1){
			act_v[i]=1;
			num_act_v++;
		}
	}

	//printf("A0: active_node %d\n",num_act_v);

	int num_act_v_prec=0;
	int count = 1;
	while(num_act_v_prec!=num_act_v){
		num_act_v_prec = num_act_v;
		
		num_act_v=0;

		hipMemcpy(d_act_v,act_v,num_v*sizeof(int),hipMemcpyHostToDevice);
		hipMemcpy(d_result,res,num_v*sizeof(int),hipMemcpyHostToDevice);

		actv<<<block,thread>>>(d_inf_v,d_trs_v,d_act_v,d_index,d_adjlist,d_result,num_v,1);

		hipMemcpy(res,d_result,num_v*sizeof(int),hipMemcpyDeviceToHost);

		for (int i = 0; i < num_v; i++)
		{
			if(res[i]==1){
				act_v[i]=1;
				num_act_v++;
			}
		}
		//printf("A%d: active_node %d\n",count,num_act_v);
		count++;
	}
	/*RESULT*/
	printf("final_active_node %d\n",num_act_v );



	hipFree(d_inf_v); hipFree(d_trs_v); hipFree(d_act_v); hipFree(d_index); hipFree(d_adjlist);hipFree(d_result);
	igraph_destroy(&graph);
	return 0;
}